#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <time.h>


#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

__global__ void sgemm_smem(int M, int N, int K, float alpha, float *A, const float *B, float beta, float *C){
  const int BLOCKSIZE = 32;
  const uint threadRow = threadIdx.y; // in the block (up to down)
  const uint threadCol = threadIdx.x; // in the block (left to right)
  const int row = blockIdx.y * BLOCKSIZE + threadRow; // global in the C matrix
  const int col = blockIdx.x * BLOCKSIZE + threadCol; // global in the C matrix
  
  // out of bounds of c matrix
  if (row >= M || col >= N){
    return;
  }

  // each block gets an equal share of shared memory
  __shared__ float As[BLOCKSIZE][BLOCKSIZE]; // 32 x 32 
  __shared__ float Bs[BLOCKSIZE][BLOCKSIZE]; // 32 x 32


  float tmp = 0.0;
  // on thread level, fill in shared memory from the corresponding A matrix or B matrix
  // on block level, loop through all blocks
  for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE) {
    if (bkIdx + threadCol < K) {
      As[threadRow][threadCol] = A[row * K + bkIdx + threadCol];
    }
    // else {
    //   As[threadRow][threadCol] = 0.0f;
    // }
    
    if (bkIdx + threadRow < K) {
      Bs[threadRow][threadCol] = B[(bkIdx + threadRow) * N + col];
    } 
    // else {
    //   Bs[threadRow][threadCol] = 0.0f;
    // }

    // wait for all threads to load their data into shared memory
    __syncthreads();

    // compute dot product for this tile
    for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
      tmp += As[threadRow][dotIdx] * Bs[dotIdx][threadCol];
      // As -> left to right
      // Bs -> top to down
    }
    
    // wait for all threads to finish using the shared memory before loading next tile
    __syncthreads();
  }
  
  // write result to global memory
  // thread level -> all threads literally have computed their dot product for that entry
  // C = alpha * (A × B) + beta * C
  C[row * N + col] = alpha * tmp + beta * C[row * N + col];
}


void sgemm_cpu(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
  // Compute C = alpha * A * B + beta * C
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      float sum = 0.0f;
      for (int k = 0; k < K; k++) {
        sum += A[i * K + k] * B[k * N + j];
      }
      C[i * N + j] = alpha * sum + beta * C[i * N + j];
    }
  }
}



// Function to verify the results between CPU and GPU
int verify_results(float *cpu_C, float *gpu_C, int M, int N) {
  const float epsilon = 1e-2; // Tolerance for floating point comparison
  int errors = 0;
  
  for (int i = 0; i < M * N; i++) {
    float diff = fabsf(cpu_C[i] - gpu_C[i]);
    if (diff > epsilon) {
      errors++;
      if (errors <= 10) { // Print only the first 10 errors
        printf("Error at index %d: GPU = %f, CPU = %f (diff = %f)\n", 
               i, gpu_C[i], cpu_C[i], diff);
      }
    }
  }
  
  if (errors > 0) {
    printf("Verification FAILED: %d errors found out of %d elements\n", errors, M * N);
    return 0;
  } else {
    printf("Verification PASSED: All values match within epsilon %e\n", epsilon);
    return 1;
  }
}

// Utility function to check CUDA errors
#define CHECK_CUDA_ERROR(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", \
                hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)


void init_matrix(float *mat, int rows, int cols){
  for (int i = 0; i < rows * cols; ++i){
    mat[i] = (float)rand() / RAND_MAX;
  }
}


int main(){
  // initialize matrix dimensions
  int M = 1024;
  int N = 1024;
  int K = 1024;


  printf("benchmarking SGEMM\n");

  // allocate host device pointers
  float *h_A = (float*)malloc(M * K * sizeof(float));
  float *h_B = (float*)malloc(K * N * sizeof(float));
  float *h_C = (float*)malloc(M * N * sizeof(float));
  float *h_C_ref = (float *)malloc(M * N * sizeof(float));
  
  srand(42); // initialize matrices
  init_matrix(h_A, M, K);
  init_matrix(h_B, K, N);
  init_matrix(h_C, M, N);

  memcpy(h_C_ref, h_C, M * N * sizeof(float)); // make a copy to the reference pointer
  float *d_A, *d_B, *d_C;
  // allocate to the device
  CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));
  // copy to the device from host
  CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));
  
  float alpha = 1.1f;
  float beta = 1.2f;

  // initialize timing
  hipEvent_t start, stop;
  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  // grid and block dimensions - 2d and 2d
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
  dim3 blockDim(32, 32);
  const int num_iterations = 10;
  
  // benchmark loop  
  CHECK_CUDA_ERROR(hipEventRecord(start));
  for (int i = 0; i < num_iterations; ++i){
    // Reset C to original values before each iteration
    CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));
    sgemm_smem<<<gridDim, blockDim>>>(M,N,K, alpha, d_A, d_B, beta, d_C); // CORRECT
  }
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));

  // calculate the statistics and print the statistics
  float milliseconds = 0;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
  float avg_time_ms = milliseconds / num_iterations;
  double gflops = (2.0 * M * N * K) / (avg_time_ms * 1e6);
  printf("average kernel execution time: %.3f ms\n", avg_time_ms);
  printf("performance: %.2f GFLOPS\n", gflops);

  CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
  
  // verify results
  sgemm_cpu(M, N, K, alpha, h_A, h_B, beta, h_C_ref);
  verify_results(h_C_ref, h_C, M, N);
  
  // clean up variables
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));
  CHECK_CUDA_ERROR(hipFree(d_A));
  CHECK_CUDA_ERROR(hipFree(d_B));
  CHECK_CUDA_ERROR(hipFree(d_C));
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_ref);
}