#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <time.h>


#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

__global__ void sgemm_naive(int M, int N, int K, float alpha, float *A, const float *B, float beta, float *C){
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  // Check if the thread is in bounds for our matrix
  if (x < M && y < N){
    float tmp = 0.0;
    for (int i = 0; i < K; ++i){
      tmp += A[x * K + i] * B[i * N + y];
    }
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }
}


void sgemm_cpu(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
  // Compute C = alpha * A * B + beta * C
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      float sum = 0.0f;
      for (int k = 0; k < K; k++) {
        sum += A[i * K + k] * B[k * N + j];
      }
      C[i * N + j] = alpha * sum + beta * C[i * N + j];
    }
  }
}

// Function to verify the results between CPU and GPU
int verify_results(float *cpu_C, float *gpu_C, int M, int N) {
  const float epsilon = 1e-2; // Tolerance for floating point comparison
  int errors = 0;
  
  for (int i = 0; i < M * N; i++) {
    float diff = fabsf(cpu_C[i] - gpu_C[i]);
    if (diff > epsilon) {
      errors++;
      if (errors <= 10) { // Print only the first 10 errors
        printf("Error at index %d: GPU = %f, CPU = %f (diff = %f)\n", 
               i, gpu_C[i], cpu_C[i], diff);
      }
    }
  }
  
  if (errors > 0) {
    printf("Verification FAILED: %d errors found out of %d elements\n", errors, M * N);
    return 0;
  } else {
    printf("Verification PASSED: All values match within epsilon %e\n", epsilon);
    return 1;
  }
}

// Utility function to check CUDA errors
#define CHECK_CUDA_ERROR(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", \
                hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)


void init_matrix(float *mat, int rows, int cols){
  for (int i = 0; i < rows * cols; ++i){
    mat[i] = (float)rand() / RAND_MAX;
  }
}


int main(){
  // initialize matrix dimensions
  int M = 1024;
  int N = 1024;
  int K = 1024;


  printf("benchmarking SGEMM\n");

  // allocate host device pointers
  float *h_A = (float*)malloc(M * K * sizeof(float));
  float *h_B = (float*)malloc(K * N * sizeof(float));
  float *h_C = (float*)malloc(M * N * sizeof(float));
  float *h_C_ref = (float *)malloc(M * N * sizeof(float));
  
  srand(42); // initialize matrices
  init_matrix(h_A, M, K);
  init_matrix(h_B, K, N);
  init_matrix(h_C, M, N);

  memcpy(h_C_ref, h_C, M * N * sizeof(float)); // make a copy to the reference pointer
  float *d_A, *d_B, *d_C;
  // allocate to the device
  CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));
  // copy to the device from host
  CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));
  
  float alpha = 1.1f;
  float beta = 1.2f;

  // initialize timing
  hipEvent_t start, stop;
  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  // grid and block dimensions - 2d and 2d
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
  dim3 blockDim(32, 32, 1);
  const int num_iterations = 10;
  
  // benchmark loop  
  CHECK_CUDA_ERROR(hipEventRecord(start));
  for (int i = 0; i < num_iterations; ++i){
    // Reset C to original values before each iteration
    CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));
    sgemm_naive<<<gridDim, blockDim>>>(M,N,K, alpha, d_A, d_B, beta, d_C); // CORRECT
  }
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));

  // calculate the statistics and print the statistics
  float milliseconds = 0;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
  float avg_time_ms = milliseconds / num_iterations;
  double gflops = (2.0 * M * N * K) / (avg_time_ms * 1e6);
  printf("average kernel execution time: %.3f ms\n", avg_time_ms);
  printf("performance: %.2f GFLOPS\n", gflops);

  CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
  
  // verify results
  sgemm_cpu(M, N, K, alpha, h_A, h_B, beta, h_C_ref);
  verify_results(h_C_ref, h_C, M, N);
  
  // clean up variables
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));
  CHECK_CUDA_ERROR(hipFree(d_A));
  CHECK_CUDA_ERROR(hipFree(d_B));
  CHECK_CUDA_ERROR(hipFree(d_C));
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_ref);
}